#include "hip/hip_runtime.h"
#include <stdio.h>
#include "timer.h"
#include <math.h>
#define PG_SIZE (4096)
#define BYTE_SIZE (256)
#define BLOCK_SIZE 32

//각 block이 공유하는 shared memory


__global__
void similarity(int data_size, char *data_old, char *data_new, int* sim)
{
  //dynamic shared memory
  int gthread_idx = blockIdx.x*blockDim.x + threadIdx.x;
 // int lthread_idx = threadIdx.x;
  int size = PG_SIZE; //각 쓰레드가 담당할 데이터 영역 크기
  int gbuf_idx = gthread_idx * size;
  int i;
  //init shared variable
  int sim_cnt=0;

  for(i=gbuf_idx; i<gbuf_idx+size && i<data_size; i++)
  {
    //similarity calculation
    sim_cnt += (data_old[i] == data_new[i]);
   }
   sim[gthread_idx] = sim_cnt;
   //printf("%d %d\n", lthread_idx, sim_cnt);
}

int main(void)
{
  int data_size = 1<<30;//1<<30;  //1GB
  char *data_old, *data_new;
  char *d_do, *d_dn;
  int num_threads;
  int num_blocks;
  int *sim, *d_sim;
  double start, finish;
  //int i;
  //데이터 초기화
  data_old = (char*)malloc(data_size*sizeof(char)); //1GB
  data_new = (char*)malloc(data_size*sizeof(char)); //1GB
  sim = (int*)malloc((data_size/PG_SIZE)*sizeof(int)); //256K*12B = 3KB

  //디바이스 데이터 할당
  hipMalloc(&d_do, data_size*sizeof(char));  //1GB
  hipMalloc(&d_dn, data_size*sizeof(char));  //1GB
  hipMalloc(&d_sim, (data_size/PG_SIZE) * sizeof(int)); //3KB

  //데이터 초기화
  for (int i = 0; i < data_size; i++) {
    data_old[i] = 1;
    data_new[i] = 1;
  }

  printf("Start Evaluation\n");
  printf("1. memcpy\n");

  GET_TIME(start);
  hipMemcpy(d_do, data_old, data_size*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(d_dn, data_new, data_size*sizeof(char), hipMemcpyHostToDevice);

  //block size 설정 (조정가능)
  num_threads = 32;
  num_blocks = data_size/PG_SIZE/num_threads; //1GB/32?? 상관무
  printf("nthread, nblock: %d %d\n", num_threads, num_blocks);
  // Perform sim_ent function on 1GB elements
  //3번째인자는 shared memory의 크기 = 516Bytes * 32 = 16KB (L1cache size 48KB)
  similarity<<<num_blocks, num_threads>>>(data_size, d_do, d_dn, d_sim);

  hipMemcpy(sim, d_sim, (data_size/PG_SIZE) * sizeof(int), hipMemcpyDeviceToHost);
  
 // for(int i=0; i<data_size/PG_SIZE; i++)
 //   printf("sim: %d\n", sim[i]);
  GET_TIME(finish);

  printf("Elapsed time = %e seconds\n", finish - start);

  hipFree(d_do);
  hipFree(d_dn);
  hipFree(d_sim);
  
  free(data_old);
  free(data_new);
  free(sim);
}